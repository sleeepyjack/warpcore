#include "hip/hip_runtime.h"
#include <iostream>
#include <warpcore/bloom_filter.cuh>
#include <helpers/timers.cuh>

double binom(
    std::uint64_t n,
    std::uint64_t k,
    double p)
{
    double res = 1.0;

    for(std::uint64_t i = n - k + 1; i <= n; ++i)
    {
        res = res * i;
    }

    for(std::uint64_t i = 1; i <= k; ++i)
    {
        res = res / i;
    }

    res = res * pow(p, k) * pow(1.0 - p, n - k);

    return res;
}

double fpr_std(
    std::uint64_t m,
    std::uint64_t n,
    std::uint64_t k)
{
    return std::pow(1.0-std::pow(1.0-1.0/m, n*k), k);
}

// according to http://algo2.iti.kit.edu/documents/cacheefficientbloomfilters-jea.pdf
double fpr_block(
    std::uint64_t m,
    std::uint64_t n,
    std::uint64_t k,
    std::uint64_t b)
{
    double res = 0.0;

    #pragma omp parallel for reduction(+:res)
    for(std::uint64_t i = 0; i < 5*n/b; ++i)
    {
        res += binom(n, i, 1.0/double(b)) * fpr_std(m/b, i, k);
    }

    return res;
}

int main(int argc, char *argv[])
{
    using data_t   = std::uint64_t;
    using slot_t = std::uint64_t;
    using index_t = std::uint64_t;
    using hasher_t = warpcore::hashers::MurmurHash<data_t>;

    static constexpr std::uint64_t seed = 42;
    std::uint64_t n = 1ULL << 26;
    std::uint64_t m = 1ULL << 33;
    std::uint64_t k = 6;
    static constexpr std::uint64_t cg_size = 1;
    static constexpr std::uint64_t block_bits = sizeof(slot_t)*8*cg_size;

    if(argc > 1)
    {
        if(argc != 4)
        {
            std::cerr << "invalid argument" << std::endl;
            return -1;
        }

        n = std::stoull(argv[1]);
        m = std::stoull(argv[2]);
        k = std::stoull(argv[3]);
    }

    using filter_t = warpcore::BloomFilter<
        data_t,
        hasher_t,
        slot_t,
        cg_size>;

    filter_t filter(m, k, seed);

    data_t* tp_data_h; hipHostMalloc(&tp_data_h, sizeof(data_t)*n); CUERR
    data_t* fp_data_h; hipHostMalloc(&fp_data_h, sizeof(data_t)*n); CUERR
    data_t* tp_data_d; hipMalloc(&tp_data_d, sizeof(data_t)*n); CUERR
    data_t* fp_data_d; hipMalloc(&fp_data_d, sizeof(data_t)*n); CUERR

    bool* flags_h; hipHostMalloc(&flags_h, sizeof(bool)*n); CUERR
    bool* flags_d; hipMalloc(&flags_d, sizeof(bool)*n); CUERR

    #pragma omp parallel for
    for(index_t i = 0; i < n; i++)
    {
        tp_data_h[i] = i+1;
        fp_data_h[i] = n+i+1;
    }

    hipMemcpy(tp_data_d, tp_data_h, sizeof(data_t)*n, H2D); CUERR
    hipMemcpy(fp_data_d, fp_data_h, sizeof(data_t)*n, H2D); CUERR
    hipMemset(flags_d, 0, sizeof(bool)*n); CUERR

    std::cout
    << "n=" << n
    << "\tm=" << m
    << "\tk=" << k
    << "\tcg=" << cg_size << std::endl;

    {
        helpers::GpuTimer timer("insert");
        filter.insert(tp_data_d, n);
    }

    {
        helpers::GpuTimer timer("retrieve_tp");
        filter.retrieve(tp_data_d, n, flags_d);
    }

    hipMemcpy(flags_h, flags_d, sizeof(bool)*n, D2H); CUERR

    std::uint64_t tp = 0;
    #pragma omp parallel for reduction(+:tp)
    for(index_t i = 0; i < n; i++)
    {
        if(flags_h[i])
        {
            tp++;
        }
    }

    hipMemset(flags_d, 0, sizeof(bool)*n); CUERR

    {
        helpers::GpuTimer timer("retrieve_fp");
        filter.retrieve(fp_data_d, n, flags_d);
    }

    hipMemcpy(flags_h, flags_d, sizeof(bool)*n, D2H); CUERR

    std::uint64_t fp = 0;
    #pragma omp parallel for reduction(+:fp)
    for(index_t i = 0; i < n; i++)
    {
        if(flags_h[i])
        {
            fp++;
        }
    }

    std::string test_result = (tp == n) ? "pass" : "fail";
    test_result += " (" + std::to_string(tp) + "/" + std::to_string(n) + ")";
    std::cout << "test: " << test_result << std::endl;

    std::cout
    << "fpr: true=" << float(fp)/float(n)
    << " (" << std::to_string(fp) << "/" << std::to_string(n) << ")"
    << "\tstd=" << fpr_std(m, n, k)
    << "\tblock=" << fpr_block(m, n, k, m/block_bits)
    << "\tmember=" << filter.fpr(n) << std::endl << std::endl;

    hipHostFree(tp_data_h);
    hipFree(tp_data_d);
    hipHostFree(fp_data_h);
    hipFree(fp_data_d);
    hipHostFree(flags_h);
    hipFree(flags_d);

    hipDeviceSynchronize(); CUERR
}
