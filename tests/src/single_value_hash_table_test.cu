#include "hip/hip_runtime.h"
#include <catch2/catch_all.hpp>
#include <warpcore/single_value_hash_table.cuh>

TEMPLATE_TEST_CASE_SIG(
    "SingleValueHashTable",
    "[singlevalue][hashtable][template]",
    ((class Key, class Value), Key, Value),
    (std::uint32_t, std::uint32_t),
    (std::uint32_t, std::uint64_t),
    (std::uint64_t, std::uint32_t),
    (std::uint64_t, double))
{
    using namespace warpcore;

    using probing_scheme_t = defaults::probing_scheme_t<Key, 8>;

    using hash_table_t =
        SingleValueHashTable<
            Key, Value,
            defaults::empty_key<Key>(),
            defaults::tombstone_key<Key>(),
            probing_scheme_t,
            defaults::table_storage_t<Key, Value>,
            defaults::temp_memory_bytes()>;

    const index_t min_capacity = GENERATE(as<index_t>{}, 12345, 4242424, 696969);
    const index_t valid_capacity =
        warpcore::detail::get_valid_capacity(min_capacity, probing_scheme_t::cg_size());
    const float load = GENERATE(as<float>{}, 0.5, 0.7, 0.8);
    const Key seed = GENERATE(as<Key>{}, 5, 42);
    const index_t n = float(valid_capacity) * load;

    CAPTURE(min_capacity, valid_capacity, load, seed, n);

    hash_table_t hash_table(min_capacity); CUERR
    REQUIRE(hash_table.peek_status() == Status::none());
    REQUIRE(hipPeekAtLastError() == hipSuccess);

    SECTION("state after object creation")
    {
        CHECK(hash_table.size() == 0);
        CHECK(hash_table.capacity() == valid_capacity);
        CHECK(hash_table.capacity() >= min_capacity);
        CHECK(hash_table.peek_status() == Status::none());
        CHECK(hipPeekAtLastError() == hipSuccess);
    }

    Key* keys_in_d = nullptr;
    hipMalloc(&keys_in_d, sizeof(Key)*n);
    Key* keys_out_d = nullptr;
    hipMalloc(&keys_out_d, sizeof(Key)*n);
    Value* values_in_d = nullptr;
    hipMalloc(&values_in_d, sizeof(Value)*n);
    Value* values_out_d = nullptr;
    hipMalloc(&values_out_d, sizeof(Value)*n);

    // generate pseudo-random unique keys and values
    helpers::lambda_kernel
    <<<SDIV(n, WARPCORE_BLOCKSIZE), WARPCORE_BLOCKSIZE>>>
    ([=] DEVICEQUALIFIER () mutable
    {
        const index_t tid = blockDim.x * blockIdx.x + threadIdx.x;
        Key i = tid;
        if(tid < n)
        {
            Key out;
            do
            {
                out = hashers::MurmurHash<Key>::hash(i+seed);
                i += n;
            }
            while(!hash_table.is_valid_key(out));

            keys_in_d[tid] = out;
            values_in_d[tid] = out;
        }
    });
    CHECK(hipPeekAtLastError() == hipSuccess);

    hash_table.insert(keys_in_d, values_in_d, n);

    CHECK(hash_table.size() == n);
    CHECK(hash_table.peek_status() == Status::none());
    CHECK(hipPeekAtLastError() == hipSuccess); CUERR

    hipMemset(values_out_d, 0, sizeof(Value)*n); CUERR

    hash_table.retrieve(keys_in_d, n, values_out_d);

    CHECK(hash_table.peek_status() == Status::none());
    CHECK(hipPeekAtLastError() == hipSuccess); CUERR

    SECTION("retrieve inserted values")
    {
        hipMemset(values_out_d, 0, sizeof(Value)*n);

        hash_table.retrieve(keys_in_d, n, values_out_d);

        CHECK(hash_table.pop_status() == Status::none());

        index_t errors_h = 0;
        index_t * errors_d = nullptr;
        hipMalloc(&errors_d, sizeof(index_t));
        hipMemset(errors_d, 0, sizeof(index_t));

        helpers::lambda_kernel
        <<<SDIV(n, WARPCORE_BLOCKSIZE), WARPCORE_BLOCKSIZE>>>
        ([=] DEVICEQUALIFIER () mutable
        {
            const index_t tid = blockDim.x * blockIdx.x + threadIdx.x;
            if(tid < n)
            {
                if(values_out_d[tid] != Value(keys_in_d[tid]))
                {
                    atomicAdd(errors_d, 1);
                }
            }
        });

        hipMemcpy(&errors_h, errors_d, sizeof(index_t), D2H);

        hipFree(errors_d);

        CHECK(errors_h == 0);
        CHECK(hipPeekAtLastError() == hipSuccess);
    }

    SECTION("retrieve all")
    {
        // TODO
    }

    SECTION("erase key/value pairs")
    {
        // TODO
    }

    SECTION("for each")
    {
        std::uint64_t * num_elems = nullptr;
        hipMallocManaged(&num_elems, sizeof(std::uint64_t));
        *num_elems = 0;

        hash_table.for_each(
            [=] DEVICEQUALIFIER (Key key, const Value& value)
            {
                if(hash_table_t::is_valid_key(key))
                {
                    atomicAdd(num_elems, 1);
                }
            });

        CHECK(*num_elems == hash_table.size());
    }

    hipFree(keys_in_d);
    hipFree(keys_out_d);
    hipFree(values_in_d);
    hipFree(values_out_d);

    CHECK(hipGetLastError() == hipSuccess);
}